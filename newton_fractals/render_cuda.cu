#include "hip/hip_runtime.h"
#include "render_cuda.h"
#include "newton_fractal.h"
#include <complex>
#include <cuda/std/complex>
#include <chrono>
#include <iostream>

// Polynomial and derivative for Newton's method (example: z^3 - 1)
__device__ static inline cuda::std::complex<float> polynomial(cuda::std::complex<float> z) {
    return z * z * z - cuda::std::complex<float>(1.0f, 0.0f);
}

__device__ static inline cuda::std::complex<float> derivative(cuda::std::complex<float> z) {
    return 3.0f * z * z;
}

__device__ int newtonGpu(cuda::std::complex<float>& z) {
    for (int i = 0; i < MAX_ITERATIONS; ++i) {
        const cuda::std::complex<float> fz = polynomial(z);
        const cuda::std::complex<float> fzPrime = derivative(z);

        // Avoid division by a very small number (derivative close to zero)
        if (cuda::std::abs(fzPrime) < EPSILON)
            return MAX_ITERATIONS;

        const cuda::std::complex<float> nextZ = z - fz / fzPrime;

        if (cuda::std::abs(nextZ - z) < EPSILON) // Converged to a root
            return i;

        z = nextZ;
    }
    return MAX_ITERATIONS; // Did not converge within the maximum number of iterations
}

__global__ void renderCudaKernel(uint32_t *d_pixelBuffer, int screenWidth, int screenHeight, float xLowerBound, float yLowerBound, float xScale, float yScale) {

    int i = threadIdx.y + blockIdx.y * blockDim.y;
    int k = threadIdx.x + blockIdx.x * blockDim.x;

    if(i < screenHeight && k < screenWidth) {

        // Map pixel to complex plane
        cuda::std::complex<float> z(xLowerBound + k * xScale, yLowerBound + i * yScale);

        // Compute Newton
        const int j = newtonGpu(z);

        // Assign color based on the root and iteration count
        uint32_t brightness = 255.0f * max(0.1f, 1.0f - (float)j / MAX_ITERATIONS);

        if (j < MAX_ITERATIONS) {
            
            if (cuda::std::abs(z - cuda::std::complex<float>(1, 0)) < EPSILON)
                brightness <<= 24; // Red for Root 1
            
            else if (cuda::std::abs(z - cuda::std::complex<float>(-0.5, sqrt(3.0) / 2)) < EPSILON)
                brightness <<= 16; // Green for Root 2
            
            else if (cuda::std::abs(z - cuda::std::complex<float>(-0.5, -sqrt(3.0) / 2)) < EPSILON)
                brightness <<= 8; // Blue for Root 3
        }

        // Store color in the line buffer
        d_pixelBuffer[i * screenWidth + k] = brightness | 0x000000FF;
    }
}

void renderCuda(std::vector<uint32_t>& pixelBuffer, int screenWidth, int screenHeight, float xLowerBound, float yLowerBound, float xScale, float yScale) {
    
    uint32_t *d_pixelBuffer;
    hipMalloc(&d_pixelBuffer, sizeof(uint32_t) * screenHeight * screenWidth);

    dim3 threadsPerBlock(16, 16, 1);
    dim3 blocksPerGrid((screenWidth + threadsPerBlock.x - 1) / threadsPerBlock.x,
                       (screenHeight + threadsPerBlock.y - 1) / threadsPerBlock.y,
                       1);

    std::chrono::steady_clock::time_point begin = std::chrono::steady_clock::now();

    renderCudaKernel<<<blocksPerGrid, threadsPerBlock>>>(d_pixelBuffer, screenWidth, screenHeight, xLowerBound, yLowerBound, xScale, yScale);
    hipDeviceSynchronize();

    std::chrono::steady_clock::time_point end = std::chrono::steady_clock::now();
    std::cout << "Frame Time: " << std::chrono::duration_cast<std::chrono::microseconds>(end - begin).count() << " us\n";

    hipMemcpy(&pixelBuffer.at(0), d_pixelBuffer, sizeof(uint32_t) * screenHeight * screenWidth, hipMemcpyDeviceToHost);
    hipFree(d_pixelBuffer);
}
