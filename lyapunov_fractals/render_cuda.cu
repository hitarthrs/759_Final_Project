#include "hip/hip_runtime.h"
#include "render_cuda.h"
#include <cuda/std/cmath>
#include <iostream>
#include <chrono>

// Computes the Lyapunov exponent
__device__ float computeLyapunovDevice(char* sequence, int seqLength, float a, float b) {
    if (seqLength == 0) return -1.0f; // Invalid sequence

    float x = 0.5f; // Initial condition
    float lyapunovExponent = 0.0f;

    for (int i = 0; i < 6000; ++i) { // MAX_ITERATIONS is set to 6000
        float r = (sequence[i % seqLength] == 'A') ? a : b;

        x = r * x * (1.0f - x);
        if (x <= 0.0f || x >= 1.0f) return -1.0f;

        float derivative = cuda::std::abs(r * (1.0f - 2.0f * x));
        if (derivative < 1e-6f) return -1.0f; // Avoid log(0)

        lyapunovExponent += cuda::std::log(derivative);
    }

    return lyapunovExponent / 6000; // MAX_ITERATIONS
}

// Maps a Lyapunov exponent value to a color (RGBA)
__device__ uint32_t mapLyapunovToColorDevice(float lyapunov) {
    if (lyapunov < 0) {
        // Dark blue color for divergent points
        uint8_t intensity = static_cast<uint8_t>(128 * max(0.0f, 1.0f - cuda::std::abs(lyapunov / 1.5f)));
        return (intensity << 16) | (intensity << 8) | 64 | 0xFF; // Dark blue
    } else {
        // Dark gold color for stable points
        return 0xB8860BFF; // Dark gold (#B8860B)
    }
}

__global__ void renderCudaKernel(uint32_t *d_pixelBuffer, int screenWidth, int screenHeight, float aMin, float bMin, float aScale, float bScale, char* d_sequence, int seqLength) {

    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;

    float a = aMin + x * aScale;
    float b = bMin + y * bScale;

    if(x < screenWidth && y < screenHeight) {

        // Compute Lyapunov exponent
        float lyapunov = computeLyapunovDevice(d_sequence, seqLength, a, b);

        // Map Lyapunov exponent to color
        d_pixelBuffer[y * screenWidth + x] = mapLyapunovToColorDevice(lyapunov);
    }
}

void renderCuda(std::vector<uint32_t>& pixelBuffer, int screenWidth, int screenHeight, float aMin, float bMin, float aScale, float bScale, std::string sequence) {

    uint32_t *d_pixelBuffer;
    hipMalloc(&d_pixelBuffer, sizeof(uint32_t) * screenHeight * screenWidth);

    char* d_sequence;
    int seqLength = sequence.length();
    hipMalloc(&d_sequence, seqLength);
    hipMemcpy(d_sequence, &sequence.at(0), seqLength, hipMemcpyHostToDevice);

    dim3 threadsPerBlock(16, 16, 1);
    dim3 blocksPerGrid((screenWidth + threadsPerBlock.x - 1) / threadsPerBlock.x,
                       (screenHeight + threadsPerBlock.y - 1) / threadsPerBlock.y,
                       1);

    // Start timer
    auto startTime = std::chrono::high_resolution_clock::now();

    renderCudaKernel<<<blocksPerGrid, threadsPerBlock>>>(d_pixelBuffer, screenWidth, screenHeight, aMin, bMin, aScale, bScale, d_sequence, seqLength);
    hipDeviceSynchronize();

    // Stop timer
    auto endTime = std::chrono::high_resolution_clock::now();
    auto duration = std::chrono::duration_cast<std::chrono::milliseconds>(endTime - startTime).count();
    std::cout << "Fractal computed in " << duration << " ms\n";

    hipMemcpy(&pixelBuffer.at(0), d_pixelBuffer, sizeof(uint32_t) * screenHeight * screenWidth, hipMemcpyDeviceToHost);
    hipFree(d_pixelBuffer);
    hipFree(d_sequence);
}
