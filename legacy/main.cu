#include "hip/hip_runtime.h"
#define SDL_MAIN_HANDLED
#include <SDL.h>
#include <stdio.h>
#include <chrono>
#include "gpu.h"

#define SCREEN_WIDTH 1920
#define SCREEN_HEIGHT 1200
__device__ int maxIter = 100;
//__device__ double bounds[4];

int mandel(float cr, float ci, int maxIter){

    // Temp variables to iterate
    int i;
    float zr = 0;
    float zi = 0;
    float zrsqr = 0;
    float zisqr = 0;

    // Mandelbrot Set determination algorithm
    for(i = 0; i < maxIter; i++) {
        zi = (2 * zi * zr) + ci;
        zr = zrsqr - zisqr + cr;
        zrsqr = zr * zr;
        zisqr = zi * zi;

        if(zrsqr + zisqr > 4.0){
            break;
        }
    }

    return i;
}

void reframe(float zoomRatio,
             int x, int y, int w, int h,
             double *xScale, double *yScale,
             double *xLowerBound, double *xUpperBound,
             double *yLowerBound, double *yUpperBound){

    // Relative mouse position in window
    double xRatio = (double)x/(double)w;
    double yRatio = (double)y/(double)h;

    // Recalculate bounds
    double xLowerBoundNext = *xLowerBound + ((double)zoomRatio * xRatio * (*xUpperBound - *xLowerBound));
    *xUpperBound = *xUpperBound - ((double)zoomRatio * (1.0 - xRatio) * (*xUpperBound - *xLowerBound));
    *xLowerBound = xLowerBoundNext;
    double yLowerBoundNext = *yLowerBound + ((double)zoomRatio * yRatio * (*yUpperBound - *yLowerBound));
    *yUpperBound = *yUpperBound - ((double)zoomRatio * (1.0 - yRatio) * (*yUpperBound - *yLowerBound));
    *yLowerBound = yLowerBoundNext;

    // Recalculate step size
    *xScale = (*xUpperBound - *xLowerBound) / w;
    *yScale = (*yUpperBound - *yLowerBound) / h;

    return;
}

__global__ void kernel(uint32_t* buf, double* bounds){

    // Pixel this thread is responsible for
    int x = blockDim.x * blockIdx.x + threadIdx.x;
    int y = blockDim.y * blockIdx.y + threadIdx.y;

    // Frame overrun guard
    if(x < SCREEN_WIDTH && y < SCREEN_HEIGHT){

        unsigned int pos = SCREEN_WIDTH * y + x;

        // Complex number to calculate
        double cr = bounds[0] + x * bounds[2];
        double ci = bounds[1] + y * bounds[3];

        // Temp variables to iterate
        int i;
        float zr = 0;
        float zi = 0;
        float zrsqr = 0;
        float zisqr = 0;

        // Mandelbrot Set determination algorithm
        for(i = 0; i < maxIter; i++) {
            zi = (2 * zi * zr) + ci;
            zr = zrsqr - zisqr + cr;
            zrsqr = zr * zr;
            zisqr = zi * zi;

            if(zrsqr + zisqr > 4.0){
                break;
            }
        }

        // Store color to buffer
        if(i >= maxIter){
            buf[pos] = 0;
        }else{
            buf[pos] = 256*256*lround(32.0*log2((double)(i+1)));
        }
    }
}

// ------------- MAIN ---------------
int main(){

    // Initialize SDL, window, screen, renderer, and texture

    if(SDL_Init(SDL_INIT_VIDEO) < 0){
        printf("Error: SDL failed to initialize\nSDL Error: '%s'\n", SDL_GetError());
        return 1;
    }

    SDL_Window *window = SDL_CreateWindow("Mandelbrot", SDL_WINDOWPOS_CENTERED, SDL_WINDOWPOS_CENTERED, SCREEN_WIDTH, SCREEN_HEIGHT, 0);
    if(!window){
        printf("Error: Failed to open window\nSDL Error: '%s'\n", SDL_GetError());
        return 1;
    }

    SDL_Surface *screen = SDL_CreateRGBSurface(0, SCREEN_WIDTH, SCREEN_HEIGHT, 32, 0xFF000000, 0x00FF0000, 0x0000FF00, 0x000000FF);
    if(!screen){
        printf("Error: Failed to create screen.\nSDL Error: '%s'\n", SDL_GetError());
        return 1;
    }

    SDL_Renderer *renderer = SDL_CreateRenderer(window, -1, 0);
    if(!renderer){
        printf("Error: Failed to create renderer\nSDL Error: '%s'\n", SDL_GetError());
        return 1;
    }

    SDL_Texture *texture = SDL_CreateTexture(renderer, SDL_PIXELFORMAT_RGBA8888, SDL_TEXTUREACCESS_STREAMING | SDL_TEXTUREACCESS_TARGET, SCREEN_WIDTH, SCREEN_HEIGHT);
    if(!texture){
        printf("Error: Failed to create texture.\nSDL Error: '%s'\n", SDL_GetError());
        return 1;
    }

    // Allocate bounds
    // bounds[0] = xLowerBound
    // bounds[1] = yLowerBound
    // bounds[2] = xScale
    // bounds[3] = yScale
    double* bounds;
    hipError_t err = hipMalloc(&bounds, 4*sizeof(double));
    if(err != hipSuccess){
        printf("Failed to allocate GPU memory.\n");
    }

    // Allocate maxIter
    
    // int* maxIter;
    // err = hipMalloc(&maxIter, sizeof(int));
    // if(err != hipSuccess){
    //     printf("Failed to allocate GPU memory.\n");
    // }

    // Allocate frame buffer
    uint32_t* buffer;
    err = hipMalloc(&buffer, SCREEN_WIDTH*SCREEN_HEIGHT*4);
    if(err != hipSuccess){
        printf("Failed to allocate GPU memory.\n");
    }

    int xMouse, yMouse;         // Mouse position
    int update = 1;             // Update frame
    float zoomInRatio = 0.5;    // Amount to zoom in by
    float zoomOutRatio = -1.0;  // Amount to zoom out by
    Uint32 buttons;             // buttons to parse
    const Uint8* keys = SDL_GetKeyboardState(NULL);    // keys to parse
    int cudaImp = 0;            // use cuda implementation
    int j;                      // iterator in CPU implementation

    // Default bounds
    double* boundsHost = (double*)malloc(4*sizeof(double));
    double xLowerBound = -2.21;
    double xUpperBound = 1.63;
    double yLowerBound = -1.2;
    double yUpperBound = 1.2;
    double xScale = (xUpperBound - xLowerBound) / SCREEN_WIDTH;
    double yScale = (yUpperBound - yLowerBound) / SCREEN_HEIGHT;

    // Fill bounds and send to device
    boundsHost[0] = xLowerBound;
    boundsHost[1] = yLowerBound;
    boundsHost[2] = xScale;
    boundsHost[3] = yScale;
    hipMemcpy(bounds, boundsHost, 4*sizeof(double), hipMemcpyHostToDevice);

    // Send maxIter to device
    int maxIterHost = 100;
    // *maxIterHost = 100;
    // hipMemcpy(maxIter, maxIterHost, sizeof(int), hipMemcpyHostToDevice);
    

    dim3 tpb; // Threads per block
    tpb.x = 32;
    tpb.y = 32;

    dim3 bpg; // Blocks per grid
    bpg.x = (SCREEN_WIDTH + tpb.x - 1)/tpb.x;
    bpg.y = (SCREEN_HEIGHT + tpb.y - 1)/tpb.y;

    printf("\nLaunching CPU implementation first,\n");
    printf("Press 'S' to switch between CPU and CUDA implementations.\n");

    bool running = true;
    while(running){
        SDL_Event event;
        while(SDL_PollEvent(&event)){
            switch(event.type){
                case SDL_QUIT: // Exit loop and clean up
                    running = false;
                    break;

                case SDL_MOUSEBUTTONDOWN: // Mouse button pressed

                    buttons = SDL_GetMouseState(&xMouse, &yMouse);
                    if((buttons & SDL_BUTTON_LMASK) != 0){

                        // Left click: zoom in
                        update = 1;
                        reframe(zoomInRatio,
                                xMouse, yMouse, SCREEN_WIDTH, SCREEN_HEIGHT,
                                &xScale, &yScale,
                                &xLowerBound, &xUpperBound,
                                &yLowerBound, &yUpperBound);

                    }else if((buttons & SDL_BUTTON_RMASK) != 0){

                        // Right click: zoom out
                        update = 1;
                        reframe(zoomOutRatio,
                                xMouse, yMouse, SCREEN_WIDTH, SCREEN_HEIGHT,
                                &xScale, &yScale,
                                &xLowerBound, &xUpperBound,
                                &yLowerBound, &yUpperBound);

                    }

                    // Update bounds on device
                    boundsHost[0] = xLowerBound;
                    boundsHost[1] = yLowerBound;
                    boundsHost[2] = xScale;
                    boundsHost[3] = yScale;
                    hipMemcpy(bounds, boundsHost, 4*sizeof(double), hipMemcpyHostToDevice);

                    break;

                case SDL_KEYDOWN:
                    SDL_GetKeyboardState(NULL);
                    if(keys[SDL_Scancode::SDL_SCANCODE_S]){
                        
                        update = 1;
                        if(cudaImp){
                            printf("\nSwitching to CPU implementation (Patience!)...\n");
                            cudaImp = 0;
                        }else{
                            printf("\nSwitching to CUDA implementation...\n");
                            cudaImp = 1;
                        }
                    }
                    break;

                default:
                    break;
            }
        }

        if(update){
            update = 0;

            // Time Frame-time
            std::chrono::steady_clock::time_point begin = std::chrono::steady_clock::now();

            if(cudaImp){ // Use CUDA

                // Render and retrieve frame
                SDL_LockSurface(screen);
                kernel<<<bpg, tpb>>>(buffer, bounds);
                hipMemcpy(screen->pixels, buffer, SCREEN_WIDTH*SCREEN_HEIGHT*4, hipMemcpyDeviceToHost);
                SDL_UnlockSurface(screen);

                // Update Screen
                SDL_UpdateTexture(texture, NULL, screen->pixels, screen->pitch);
                SDL_RenderCopy(renderer, texture, NULL, NULL);
                SDL_RenderPresent(renderer);

            }else{ // Use CPU

                // Iterate through all pixels
                for(int i = 0; i < SCREEN_HEIGHT; i++){
                    for(int k = 0; k < SCREEN_WIDTH; k++){

                        // Calculate if in Mandelbrot Set, set color accordingly
                        j = mandel(xLowerBound + (xScale * k), yLowerBound + (yScale * i), maxIterHost);
                        if(j >= maxIterHost){
                            SDL_SetRenderDrawColor(renderer, 0, 0, 0, SDL_ALPHA_OPAQUE);
                        }else{
                            SDL_SetRenderDrawColor(renderer, 32*log2(j+1), 0, 0, SDL_ALPHA_OPAQUE);
                        }
                        SDL_RenderDrawPoint(renderer, k, i);
                    }
                    SDL_RenderPresent(renderer); // Update screen after each line to show progress
                }
            }

            // Print Frame-time
            std::chrono::steady_clock::time_point end = std::chrono::steady_clock::now();
            printf("Frame Time: %I64i us\n", std::chrono::duration_cast<std::chrono::microseconds>(end-begin).count());
        }
    }

    // Free host and device memory
    hipFree(bounds);
    //hipFree(maxIter);
    hipFree(buffer);
    free(boundsHost);
    //free(maxIterHost);

    // Quit
    SDL_DestroyWindow(window);
    SDL_Quit();

    return 0;
}